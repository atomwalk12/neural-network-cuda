
#include <hip/hip_runtime.h>
#include <iostream>
#include <cudnn.h>

void checkCUDNN(cudnnStatus_t status) {
    if (status != CUDNN_STATUS_SUCCESS) {
        std::cerr << "Error on line " << __LINE__ << ": "
                  << cudnnGetErrorString(status) << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    cudnnHandle_t cudnn;
    cudnnStatus_t status = cudnnCreate(&cudnn);
    
    if (status == CUDNN_STATUS_SUCCESS) {
        std::cout << "cuDNN initialized successfully!" << std::endl;
        
        // Get version
        std::cout << "cuDNN version: " << cudnnGetVersion() << std::endl;
    } else {
        std::cerr << "Failed to initialize cuDNN: " 
                  << cudnnGetErrorString(status) << std::endl;
        return -1;
    }

    cudnnDestroy(cudnn);
    return 0;
}